#include "hip/hip_runtime.h"
/**
 * CUDA MD5 cracker
 * Copyright (C) 2015  Konrad Kusnierz <iryont@gmail.com>
 *
 * This program is free software; you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation; either version 2 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License along
 * with this program; if not, write to the Free Software Foundation, Inc.,
 * 51 Franklin Street, Fifth Floor, Boston, MA 02110-1301 USA.
 */

#include <stdio.h>
#include <iostream>
#include <time.h>
#include <string.h>
#include <stdlib.h>
#include <stdint.h>
#include <sstream>
#include <csignal>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hiprand/hiprand_kernel.h>

#define CONST_WORD_LIMIT 10
#define CONST_CHARSET_LIMIT 100

#define CONST_CHARSET "abcdefghijklmnopqrstuvwxyzABCDEFGHIJKLMNOPQRSTUVWXYZ0123456789"
#define CONST_CHARSET_LENGTH (sizeof(CONST_CHARSET) - 1)

#define CONST_WORD_LENGTH_MIN 1
#define CONST_WORD_LENGTH_MAX 8

#define TOTAL_BLOCKS 80UL//16384UL
#define TOTAL_THREADS 32UL
#define HASHES_PER_KERNEL 128UL

#include "assert.cu"
#include "md5.cu"

/* Global variables */
uint8_t g_wordLength;

char g_word[(CONST_WORD_LIMIT)];
char g_charset[CONST_CHARSET_LIMIT];
char g_cracked[1024][10];
uint32_t salts[1024*4];
uint32_t* hashes = new uint32_t[1024];

__device__ char g_deviceCracked[1024][10];
__device__ char g_deviceCharset[CONST_CHARSET_LIMIT];

__device__ __host__ bool next(uint8_t* length, char* word, uint32_t increment){
  uint32_t idx = 0;
  uint32_t add = 0;
  
  while(increment > 0 && idx < (CONST_WORD_LIMIT)){
    if(idx >= *length && increment > 0){
      increment--;
    }
    
    add = increment + word[idx];
    word[idx] = add % CONST_CHARSET_LENGTH;
    increment = add / CONST_CHARSET_LENGTH;
    idx++;
  }
  
  if(idx > *length){
    *length = idx;
  }
  
  if(idx > CONST_WORD_LENGTH_MAX){
    return false;
  }

  return true;
}

void md5Hash_salt(unsigned char* data, uint32_t length, uint32_t *a1, uint32_t *b1, uint32_t *c1, uint32_t *d1){
  const uint32_t a0 = 0x67452301;
  const uint32_t b0 = 0xEFCDAB89;
  const uint32_t c0 = 0x98BADCFE;
  const uint32_t d0 = 0x10325476;

  uint32_t a = 0;
  uint32_t b = 0;
  uint32_t c = 0;
  uint32_t d = 0;

  uint32_t vals[16] = {0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0};

  int i = 0;
  for(i=0; i < length; i++){
    vals[i / 4] |= data[i] << ((i % 4) * 8);
  }

  #define lin0  (vals[0])//x
  #define lin1  (vals[1])//y
  #define lin2  (vals[2])//z
  #define lin3  (vals[3])
  #define lin4  (vals[4])
  #define lin5  (vals[5])
  #define lin6  (vals[6])
  #define lin7  (vals[7])
  #define lin8  (vals[8])
  #define lin9  (vals[9])
  #define lin10 (vals[10])
  #define lin11 (vals[11])
  #define lin12 (vals[12])
  #define lin13 (vals[13])
  #define lin14 (vals[14])
  #define lin15 (vals[15])

  //Initialize hash value for this chunk:
  a = a0;
  b = b0;
  c = c0;
  d = d0;

  /* Round 1 */
  #define S11 7
  #define S12 12
  #define S13 17
  #define S14 22
  FF ( a, b, c, d, lin0,  S11, 3614090360); /* 1 */
  FF ( d, a, b, c, lin1,  S12, 3905402710); /* 2 */
  FF ( c, d, a, b, lin2,  S13,  606105819); /* 3 */
  FF ( b, c, d, a, lin3,  S14, 3250441966); /* 4 */
  FF ( a, b, c, d, lin4,  S11, 4118548399); /* 5 */
  FF ( d, a, b, c, lin5,  S12, 1200080426); /* 6 */
  FF ( c, d, a, b, lin6,  S13, 2821735955); /* 7 */
  FF ( b, c, d, a, lin7,  S14, 4249261313); /* 8 */
  FF ( a, b, c, d, lin8,  S11, 1770035416); /* 9 */
  FF ( d, a, b, c, lin9,  S12, 2336552879); /* 10 */
  FF ( c, d, a, b, lin10, S13, 4294925233); /* 11 */
  FF ( b, c, d, a, lin11, S14, 2304563134); /* 12 */
  FF ( a, b, c, d, lin12, S11, 1804603682); /* 13 */
  FF ( d, a, b, c, lin13, S12, 4254626195); /* 14 */
  FF ( c, d, a, b, lin14, S13, 2792965006); /* 15 */
  FF ( b, c, d, a, lin15, S14, 1236535329); /* 16 */

  /* Round 2 */
  #define S21 5
  #define S22 9
  #define S23 14
  #define S24 20
  GG ( a, b, c, d, lin1, S21, 4129170786); /* 17 */
  GG ( d, a, b, c, lin6, S22, 3225465664); /* 18 */
  GG ( c, d, a, b, lin11, S23,  643717713); /* 19 */
  GG ( b, c, d, a, lin0, S24, 3921069994); /* 20 */
  GG ( a, b, c, d, lin5, S21, 3593408605); /* 21 */
  GG ( d, a, b, c, lin10, S22,   38016083); /* 22 */
  GG ( c, d, a, b, lin15, S23, 3634488961); /* 23 */
  GG ( b, c, d, a, lin4, S24, 3889429448); /* 24 */
  GG ( a, b, c, d, lin9, S21,  568446438); /* 25 */
  GG ( d, a, b, c, lin14, S22, 3275163606); /* 26 */
  GG ( c, d, a, b, lin3, S23, 4107603335); /* 27 */
  GG ( b, c, d, a, lin8, S24, 1163531501); /* 28 */
  GG ( a, b, c, d, lin13, S21, 2850285829); /* 29 */
  GG ( d, a, b, c, lin2, S22, 4243563512); /* 30 */
  GG ( c, d, a, b, lin7, S23, 1735328473); /* 31 */
  GG ( b, c, d, a, lin12, S24, 2368359562); /* 32 */

  /* Round 3 */
  #define S31 4
  #define S32 11
  #define S33 16
  #define S34 23
  HH ( a, b, c, d, lin5, S31, 4294588738); /* 33 */
  HH ( d, a, b, c, lin8, S32, 2272392833); /* 34 */
  HH ( c, d, a, b, lin11, S33, 1839030562); /* 35 */
  HH ( b, c, d, a, lin14, S34, 4259657740); /* 36 */
  HH ( a, b, c, d, lin1, S31, 2763975236); /* 37 */
  HH ( d, a, b, c, lin4, S32, 1272893353); /* 38 */
  HH ( c, d, a, b, lin7, S33, 4139469664); /* 39 */
  HH ( b, c, d, a, lin10, S34, 3200236656); /* 40 */
  HH ( a, b, c, d, lin13, S31,  681279174); /* 41 */
  HH ( d, a, b, c, lin0, S32, 3936430074); /* 42 */
  HH ( c, d, a, b, lin3, S33, 3572445317); /* 43 */
  HH ( b, c, d, a, lin6, S34,   76029189); /* 44 */
  HH ( a, b, c, d, lin9, S31, 3654602809); /* 45 */
  HH ( d, a, b, c, lin12, S32, 3873151461); /* 46 */
  HH ( c, d, a, b, lin15, S33,  530742520); /* 47 */
  HH ( b, c, d, a, lin2, S34, 3299628645); /* 48 */

  /* Round 4 */
  #define S41 6
  #define S42 10
  #define S43 15
  #define S44 21
  II ( a, b, c, d, lin0, S41, 4096336452); /* 49 */
  II ( d, a, b, c, lin7, S42, 1126891415); /* 50 */
  II ( c, d, a, b, lin14, S43, 2878612391); /* 51 */
  II ( b, c, d, a, lin5, S44, 4237533241); /* 52 */
  II ( a, b, c, d, lin12, S41, 1700485571); /* 53 */
  II ( d, a, b, c, lin3, S42, 2399980690); /* 54 */
  II ( c, d, a, b, lin10, S43, 4293915773); /* 55 */
  II ( b, c, d, a, lin1, S44, 2240044497); /* 56 */
  II ( a, b, c, d, lin8, S41, 1873313359); /* 57 */
  II ( d, a, b, c, lin15, S42, 4264355552); /* 58 */
  II ( c, d, a, b, lin6, S43, 2734768916); /* 59 */
  II ( b, c, d, a, lin13, S44, 1309151649); /* 60 */
  II ( a, b, c, d, lin4, S41, 4149444226); /* 61 */
  II ( d, a, b, c, lin11, S42, 3174756917); /* 62 */
  II ( c, d, a, b, lin2, S43,  718787259); /* 63 */
  II ( b, c, d, a, lin9, S44, 3951481745); /* 64 */

  *a1 = a+a0;
  *b1 = b+b0;
  *c1 = c+c0;
  *d1 = d+d0;
}

__global__ void md5Crack(uint8_t wordLength, char* charsetWord, uint32_t hash01[], uint32_t salts[1024*4], uint32_t offset){
  uint32_t idx = (blockIdx.x * blockDim.x + threadIdx.x) * HASHES_PER_KERNEL;
  uint32_t index = (blockIdx.x/40);
  /* Shared variables */
  __shared__ char sharedCharset[CONST_CHARSET_LIMIT];
  
  /* Thread variables */
  char threadCharsetWord[CONST_WORD_LIMIT];
  char threadTextWord[CONST_WORD_LIMIT];
  uint8_t threadWordLength;
  uint32_t threadHash01, threadHash02, threadHash03, threadHash04;

  /* Copy everything to local memory */
  memcpy(threadCharsetWord, charsetWord, CONST_WORD_LIMIT);
  memcpy(&threadWordLength, &wordLength, sizeof(uint8_t));
  memcpy(sharedCharset, g_deviceCharset, sizeof(uint8_t) * CONST_CHARSET_LIMIT);
  /* Increment current word by thread index */
  next(&threadWordLength, threadCharsetWord, idx);
  for(uint32_t hash = 0; hash < HASHES_PER_KERNEL; hash++){
    for(uint32_t i = 0; i < threadWordLength; i++){
      threadTextWord[i] = sharedCharset[threadCharsetWord[i]];
    }
    md5Hash((unsigned char*)threadTextWord, threadWordLength, &threadHash01, &threadHash02, &threadHash03, &threadHash04, salts[((index*4)+(offset*64))+0], salts[((index*4)+(offset*64))+1], salts[((index*4)+(offset*64))+2], salts[((index*4)+(offset*64))+3]);
    //printf("probably illegal\n");
    if((threadHash01 & 0xFFFFFF) == hash01[index+(offset*64)]){
      memcpy(g_deviceCracked[index+(offset*64)], threadTextWord, threadWordLength);
      break;
    }
    
    if(!next(&threadWordLength, threadCharsetWord, 1)){
      break;
    }
  }
}

int main(int argc, char* argv[]){
  /* Check arguments */
  if(argc != 2){
    std::cout << argv[0] << " <infile>" << std::endl;
    return -1;
  }
  
  /* Time */
  hipEvent_t clockBegin;
  hipEvent_t clockLast;
  
  hipEventCreate(&clockBegin);
  hipEventCreate(&clockLast);
  hipEventRecord(clockBegin, 0);

  FILE* infile = fopen(argv[1], "r");
  if(infile == NULL){
    std::cout << "Could not open file " << argv[1] << std::endl;
    return -1;
  }
  char** original = (char**)malloc(sizeof(char) * 1024*6);
  //read hashes and salt from infile
  for(int i = 0; i < 1024; i++){
    char* salt = new char[64];
    char* hash = new char[6];
    fscanf(infile, "%s %s", salt, hash);
    uint32_t hash_inp = strtol(hash, NULL, 16);  
    uint32_t Hash = ((hash_inp>>24)&0xff) | // move byte 3 to byte 0
                      ((hash_inp<<8)&0xff0000) | // move byte 1 to byte 2
                      ((hash_inp>>8)&0xff00) | // move byte 2 to byte 1
                      ((hash_inp<<24)&0xff000000); // byte 0 to byte 3
    Hash >>= 8;
    hashes[i] = Hash;
    original[i] = hash;
    md5Hash_salt((unsigned char*)salt, 64, &salts[(i*4)+0], &salts[(i*4)+1], &salts[(i*4)+2], &salts[(i*4)+3]);
  }
  // close handle to file
  fclose(infile);
  
  /* Amount of available devices */
  int devices;
  ERROR_CHECK(hipGetDeviceCount(&devices));
  
  /* Sync type */
  ERROR_CHECK(hipSetDeviceFlags(hipDeviceScheduleSpin));
  
  /* Fill memory */
  memset(g_word, 0, CONST_WORD_LIMIT);
  memcpy(g_charset, CONST_CHARSET, CONST_CHARSET_LENGTH);
  
  /* Current word length = minimum word length */
  g_wordLength = CONST_WORD_LENGTH_MIN;
  
  /* Main device */
  hipSetDevice(0);

  uint32_t* DeviceHashArr;
  uint32_t* DeviceSaltArr;
  
  /* Current word is different on each device */
  char** words = new char*[devices];

  for(int device = 0; device < devices; device++){
    hipSetDevice(device);
    
    /* Copy to each device */
    ERROR_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(g_deviceCharset), g_charset, sizeof(uint8_t) * CONST_CHARSET_LIMIT, 0, hipMemcpyHostToDevice));
    ERROR_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(g_deviceCracked), g_cracked, sizeof(char) * 1024 * 10, 0, hipMemcpyHostToDevice));
    
    /* Allocate on each device */
    ERROR_CHECK(hipMalloc((void**)&words[device], sizeof(uint8_t) * (CONST_WORD_LIMIT)));
    ERROR_CHECK(hipMalloc((void**)&DeviceHashArr, sizeof(uint32_t) * (1024)));
    ERROR_CHECK(hipMalloc((void**)&DeviceSaltArr, sizeof(uint32_t) * (1024*4)));
  }

  char** foundWords = new char*[1024];
  int currentIndex = 0;
  while(true){    
    for(int device = 0; device < devices; device++){
      hipSetDevice(device);
      
      /* Copy current data */
      ERROR_CHECK(hipMemcpy(words[device], g_word, sizeof(uint8_t) * (CONST_WORD_LIMIT), hipMemcpyHostToDevice)); 
      ERROR_CHECK(hipMemcpy(DeviceHashArr, hashes, sizeof(uint32_t) * (1024), hipMemcpyHostToDevice)); 
      ERROR_CHECK(hipMemcpy(DeviceSaltArr, salts, sizeof(uint32_t) * (4096), hipMemcpyHostToDevice));
    
      /* Start kernel */
      md5Crack<<<TOTAL_BLOCKS*64, TOTAL_THREADS>>>(g_wordLength, words[device], DeviceHashArr, DeviceSaltArr, currentIndex);
      hipError_t error = hipGetLastError();
      if(error){
        std::cout << "Error: " << hipGetErrorString(error) << std::endl;
        return -1;
      }
      
      /* Global increment */
      next(&g_wordLength, g_word, TOTAL_THREADS * HASHES_PER_KERNEL * TOTAL_BLOCKS);
    }
          
    /* Synchronize now */
    hipDeviceSynchronize();
    
    /* Copy result */
    ERROR_CHECK(hipMemcpyFromSymbol(g_cracked, HIP_SYMBOL(g_deviceCracked), sizeof(char) * 1024 * 10, 0, hipMemcpyDeviceToHost)); 
    
    /* Check result */
    int found = 0;
    for(int i = 0; i < 64; i++){
      if(g_cracked[i+(currentIndex*64)][0] != 0){
        found += 1;
      }
    }
    if(found == 64){
      for(int i = 0; i < 64; i++){
        foundWords[i+(currentIndex*64)] = g_cracked[i+(currentIndex*64)];
      }
      currentIndex += 1;
    }
    if (currentIndex == (1024/64)){
      break;
    }
  }

  FILE* outfile = fopen("outfile.txt", "w+");
  for(int i = 0; i < 1024; i++){
    if(foundWords[i] != NULL){
      fprintf(outfile, "%s %s\n", original[i], foundWords[i]);
    }
  }
  fclose(outfile);
  
  for(int device = 0; device < devices; device++){
    hipSetDevice(device);
    
    /* Free on each device */
    hipFree((void**)words[device]);
  }
  
  /* Free array */
  delete[] words;
  
  /* Main device */
  hipSetDevice(0);

  
  float milliseconds = 0;
  
  hipEventRecord(clockLast, 0);
  hipEventSynchronize(clockLast);
  hipEventElapsedTime(&milliseconds, clockBegin, clockLast);
  
  std::cout << "Notice: computation time " << milliseconds << " ms" << std::endl;
  
  hipEventDestroy(clockBegin);
  hipEventDestroy(clockLast);
}
